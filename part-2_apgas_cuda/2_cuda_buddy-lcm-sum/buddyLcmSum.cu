#include "hip/hip_runtime.h"
#include "buddyKVGSum.cuh"

__device__ int gcd(int a, int b) {
    if (a == 0) {
        return b;
    } else {
        while (b != 0) {
            if (a > b) {
                a -= b;
            } else {
                b -= a;
            }
        }

        return a;
    }
}

__device__ int lcm(const int a, const int b, const int gcd) {
    return (a / gcd) * b;
}

__device__ int f(const int a, const int b, const int min_lcm) {
    return lcm(a, b, gcd(a, b)) >= min_lcm ? 1 : 0;
}

__global__ void buddyKVGSum(const Matrix a,
        const Matrix b,
        Matrix c,
        const int n,
        const int min_lcm) {
    extern __shared__ int shared_copy[];
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    if (tx == 0) {
        if (ty == 0) {
            for (int i = 0; i < n; i++) {
                shared_copy(0, i) = a(by, i);
            }
        } else if (ty == 1) {
            for (int i = 0; i < n; i++) {
                shared_copy(1, i) = b(bx, i);
            }
        }
    }

    __syncthreads();

    if (f(shared_copy(0, tx), shared_copy(1, ty), min_lcm)) {
        atomicAdd(&shared_copy[2*n], 1);
    }

    __syncthreads();

    if (tx == 0 && ty == 0) {
        c(bx, by) = shared_copy[2*n];
    }
}

void print(const Matrix a, const int n) {
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            printf("%d ", a(i, j));
        }

        printf("\n");
    }
}

void sum(const Matrix a,
        const Matrix b,
        Matrix c,
        const int n,
        const int min_lcm,
        const int input_size,
        const int verbose) {
    Matrix ad, bd, cd;
    int *nd, *min_lcmd;

    gpuErrchk(hipMalloc((void **)&ad, input_size));
    gpuErrchk(hipMalloc((void **)&bd, input_size));
    gpuErrchk(hipMalloc((void **)&cd, input_size));
    gpuErrchk(hipMalloc((void **)&nd, sizeof(int)));
    gpuErrchk(hipMalloc((void **)&min_lcmd, sizeof(int)));

    gpuErrchk(hipMemcpy(ad, a, input_size, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(bd, b, input_size, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy((void *)nd, (void *)&n, sizeof(int), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy((void *)min_lcmd, (void *)&min_lcm, sizeof(int), hipMemcpyHostToDevice));

    if (0 != (2 & verbose)) {
        print(a, n);
        printf("\n");
        print(b, n);
        printf("\n");
    }

    dim3 gsize(n, n);
    dim3 bsize(n, n);

    clock_t time = clock();
    buddyKVGSum<<<gsize, bsize, (2*n+1)*sizeof(int)>>>(ad, bd, cd, n, min_lcm);
    gpuErrchk(hipDeviceSynchronize());
    time = clock() - time;

    gpuErrchk(hipMemcpy(c, cd, input_size, hipMemcpyDeviceToHost));

    if (0 != (1 & verbose)) {
        print(c, n);
        printf("\n");
    }

    printf("Execution time: %f\n", (float) time / CLOCKS_PER_SEC);

    gpuErrchk(hipFree(ad));
    gpuErrchk(hipFree(bd));
    gpuErrchk(hipFree(cd));

    hipDeviceReset();
}

int main(int argc, char **argv) {
    if (argc < 5) {
        printf("Program must be called with at least 4 parameters.");
        return 42;
    }

    const int n = atoi(argv[1]);
    const int m = atoi(argv[2]);
    const int min_lcm = atoi(argv[3]);
    const int seed = atoi(argv[4]);
    const int verbose = argc > 5 ? atoi(argv[5]) : 0;

    const int matrix_size = n * n;
    const int input_size = matrix_size * sizeof(int);

    Matrix a = (Matrix) malloc(input_size),
           b = (Matrix) malloc(input_size),
           c = (Matrix) malloc(input_size);

    srand(seed);

    int i, j;

    for (i = 0; i < n; ++i) {
        for (j = 0; j < n; ++j) {
            a(i, j) = rand() % (m - 1) + 1;
        }
    }

    for (i = 0; i < n; ++i) {
        for (j = 0; j < n; ++j) {
            b(i, j) = rand() % (m - 1) + 1;
        }
    }

    sum(a, b, c, n, min_lcm, input_size, verbose);

    free(a);
    free(b);
    free(c);

    return EXIT_SUCCESS;
}
